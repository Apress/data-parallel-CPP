// Copyright (C) 2023 Intel Corporation

// SPDX-License-Identifier: MIT

#include <hip/hip_runtime.h>

#include <iostream>

constexpr int count = 1024 * 1024;

// BEGIN CODE SNIP
// The CUDA kernel is a separate function
__global__ void TestKernel(int* dst) {
  auto id = blockIdx.x * blockDim.x + threadIdx.x;
  dst[id] = id;
}

int main() {
  // CUDA uses device zero by default
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  std::cout << "Running on device: " << deviceProp.name << "\n";

  int* buffer = nullptr;
  hipHostMalloc(&buffer, count * sizeof(int), hipHostMallocDefault);
  hipMemset(buffer, 0, count * sizeof(int));

  TestKernel<<<count / 256, 256>>>(buffer);
  hipDeviceSynchronize();
  // ...
// END CODE SNIP

  int mismatches = 0;
  for (int i = 0; i < count; i++) {
    if (buffer[i] != i) {
      mismatches++;
    }
  }
  if (mismatches) {
    std::cout << "Found " << mismatches << " mismatches out of "
              << count << " elements.\n";
  } else {
    std::cout << "Success.\n";
  }

  hipHostFree(buffer);
  return 0;
}
